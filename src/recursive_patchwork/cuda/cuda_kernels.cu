#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <thrust/remove.h>
#include <thrust/copy.h>
#include <thrust/functional.h>
#include <thrust/tuple.h>

// CUDA kernel for 2D rotation
__global__ void rotatePointsKernel(float* x, float* y, float* z, 
                                  float cos_a, float sin_a, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float x_orig = x[idx];
        float y_orig = y[idx];
        x[idx] = x_orig * cos_a - y_orig * sin_a;
        y[idx] = x_orig * sin_a + y_orig * cos_a;
        // z remains unchanged
    }
}

// CUDA kernel for 4x4 transformation
__global__ void transformPointsKernel(float* x, float* y, float* z,
                                     float* matrix, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float x_orig = x[idx];
        float y_orig = y[idx];
        float z_orig = z[idx];
        
        // Apply 4x4 transformation matrix
        float x_new = matrix[0] * x_orig + matrix[1] * y_orig + matrix[2] * z_orig + matrix[3];
        float y_new = matrix[4] * x_orig + matrix[5] * y_orig + matrix[6] * z_orig + matrix[7];
        float z_new = matrix[8] * x_orig + matrix[9] * y_orig + matrix[10] * z_orig + matrix[11];
        float w = matrix[12] * x_orig + matrix[13] * y_orig + matrix[14] * z_orig + matrix[15];
        
        // Homogeneous division
        x[idx] = x_new / w;
        y[idx] = y_new / w;
        z[idx] = z_new / w;
    }
}

// Functor for ego vehicle filtering
struct EgoVehicleFilter {
    float radius_squared;
    
    EgoVehicleFilter(float radius) : radius_squared(radius * radius) {}
    
    __device__ bool operator()(const thrust::tuple<float, float, float>& point) const {
        float x = thrust::get<0>(point);
        float y = thrust::get<1>(point);
        float distance_squared = x * x + y * y;
        return distance_squared > radius_squared;
    }
}; 